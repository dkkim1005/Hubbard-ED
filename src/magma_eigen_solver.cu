#include "hip/hip_runtime.h"
// Copyright (c) 2020-2021 Dongkyu Kim (dkkim1005@gmail.com)

#include "../include/magma_eigen_solver.cuh"

MAGMA_DLOBPCG_WRAPPER::MAGMA_DLOBPCG_WRAPPER(const int m):
  m_(m),
  A_h_{Magma_CSR},
  A_d_{Magma_CSR},
  b_h_{Magma_CSR},
  b_d_{Magma_CSR},
  IsAllocated(false) {}

void MAGMA_DLOBPCG_WRAPPER::construct_CSR_format(int * row, int * col, double * val, const magma_queue_t & queue)
{
  if (IsAllocated)
  {
    std::cerr << "plz excute 'free' method first." << std::flush << std::endl;
    return;
  }
  // convert to the csr format
  magma_dcsrset(m_, m_, row, col, val, &A_h_, queue);
  // initialize a vector from a random starting point
  std::vector<double> rhs(m_);
  unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::mt19937 gen(seed);
  std::uniform_real_distribution<double> dist(-1.0, 1.0);
  double norm = 0;
  for (auto & item : rhs)
  {
    item = dist(gen);
    norm += item*item;
  }
  norm = std::sqrt(norm);
  for (auto & item : rhs)
    item /= norm;
  magma_dvset(m_, 1, rhs.data(), &b_h_, queue);
  // cpu --> gpu
  magma_dmtransfer(A_h_, &A_d_, Magma_CPU, Magma_DEV, queue);
  magma_dmtransfer(b_h_, &b_d_, Magma_CPU, Magma_DEV, queue);
  IsAllocated = true;
}

void MAGMA_DLOBPCG_WRAPPER::run(magma_dopts & opts, const magma_queue_t & queue)
{
  magma_d_precondsetup(A_d_, b_d_, &opts.solver_par, &opts.precond_par, queue);
  magma_dlobpcg(A_d_, &opts.solver_par, &opts.precond_par, queue);
}

void MAGMA_DLOBPCG_WRAPPER::free(const magma_queue_t & queue)
{
  if (IsAllocated == true)
  {
    magma_dmfree(&A_h_, queue);
    magma_dmfree(&b_h_, queue);
    magma_dmfree(&A_d_, queue);
    magma_dmfree(&b_d_, queue);
  }
  IsAllocated = false;
}
