#include "hip/hip_runtime.h"
// Copyright (c) 2020-2021 Dongkyu Kim (dkkim1005@gmail.com)

#include <iostream>
#include <memory>
#include <exception>
#include <fstream>
#include "./include/Hamiltonian.hpp"
#include "./include/argparse.hpp"
#include "./include/magma_eigen_solver.cuh"

Fermion::Fockstate * basis_factory(const int L, const std::vector<int> & np);
std::vector<double> generate_harmonic_potential(const int L, const double V);

int main(int argc, char * argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("L", "system size"));
  options.push_back(pair_t("np", "# of particle numbers; one elem -> fix total #, two elem -> fix # for each spin"));
  options.push_back(pair_t("t", "hopping element"));
  options.push_back(pair_t("U", "onsite interaction"));
  options.push_back(pair_t("V", "strength of the harmonic potential"));
  options.push_back(pair_t("bias", "mat <-- mat+bias*I"));
  options.push_back(pair_t("pbc", "use periodic boundary condition (true : 1 or false : 0)"));
  // env; default value
  defaults.push_back(pair_t("t", "1"));
  defaults.push_back(pair_t("bias", "0"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);
  const unsigned int L = parser.find<unsigned int>("L");
  const unsigned int nsites = L;
  const auto np = parser.mfind<int>("np");
  const double t = parser.find<double>("t"),
    U = parser.find<double>("U"),
    bias = parser.find<double>("bias");
  const bool usePBC = parser.find<bool>("pbc");

  // number basis (Fock space of fixed # of particles)
  std::unique_ptr<Fermion::Fockstate> basis_ptr(basis_factory(L, np));
  const unsigned int HilbertSize = basis_ptr->size();
  const std::vector<double> V = generate_harmonic_potential(L, parser.find<double>("V"));
  // matrix container to deploy Hubbard Hamiltonian
  EigenSparseMatrix smatrix(HilbertSize);
  // lattice information
  ChainLattice lattice(nsites, usePBC);

  std::cout << "# construct sparse matrix... (hilbert size: " << HilbertSize << ") " << std::flush;
  // parameter set of Hubbard model
  HubbardParams params;
  params.t = t;
  params.U = U;
  params.V = V;
  construct_hubbard_hamiltonian(smatrix, *basis_ptr, lattice, params, bias);
  std::cout << "done." << std::endl << std::flush;

  // Eigen sparse --> MAGMA sparse
  auto & rawData = smatrix.get_data();
  rawData.prune(1e-15);
  rawData.makeCompressed();
  int * col = rawData.innerIndexPtr(),
    * row = rawData.outerIndexPtr();
  double * val = rawData.valuePtr();
  magma_init();
  magma_dopts opts;
  magma_queue_t queue;
  magma_queue_create(0, &queue);
  MAGMA_DLOBPCG_WRAPPER solver(HilbertSize);
  solver.construct_CSR_format(row, col, val, queue);

  // ********** MAGMA-GPU eigen value solver **********

  // options for DLOBPCG solver
  opts.solver_par.solver = Magma_LOBPCG;
  opts.solver_par.num_eigenvalues = 1; // number of eigenvalues you want to compute
  opts.solver_par.ev_length = HilbertSize;
  opts.solver_par.maxiter = 1000; // max number of iterations
  opts.solver_par.verbose = true;
  opts.solver_par.rtol = 1e-7;
  opts.solver_par.restart = 8;
  // options for preconditioner
  opts.precond_par.solver = Magma_JACOBI;
  opts.precond_par.levels = 0; // ILU(0) - no fill-in
  opts.precond_par.trisolver = Magma_CUSOLVE; //exact triangular solves

  magma_deigensolverinfo_init(&opts.solver_par, queue);

  solver.run(opts, queue);

  std::cout << "--- lowest energy:" << std::endl;
  for (int i=0; i<opts.solver_par.num_eigenvalues; ++i)
    std::cout << (opts.solver_par.eigenvalues[i]-bias) << " ";
  std::cout << std::endl;

  std::vector<double> eigenvectors(HilbertSize);
  if (hipSuccess != hipMemcpy(eigenvectors.data(), opts.solver_par.eigenvectors,
      sizeof(double)*eigenvectors.size(), hipMemcpyDeviceToHost))
    std::cerr << "check hipMemcpy!" << (__LINE__-2) << std::endl;

  solver.free(queue);

  magma_queue_destroy(queue);
  magma_finalize();

  // **************************************************

  // plot density profile
  const auto density = meas_density(*basis_ptr, eigenvectors);
  const std::string fname = "density-L" + parser.find<>("L") + "U" + parser.find<>("U")
    + "V" + parser.find<>("V") + "P" + parser.find<>("pbc") + "N" + parser.find<>("np") + ".dat";
  std::ofstream wfile(fname);
  for (int i=0; i<L; ++i)
    wfile << (i+1) << " " << density[i] << std::endl;

  return 0;
}

Fermion::Fockstate * basis_factory(const int L, const std::vector<int> & np)
{
  if (np.size() == 1)
    return new Fermion::Fockstate(L, np[0]);
  else if (np.size() == 2)
    return new Fermion::Fockstate(L, np[0], np[1]);
  else
    throw std::invalid_argument("Check the size of'np' container.");
}

std::vector<double> generate_harmonic_potential(const int L, const double V)
{
  std::vector<double> tmp(L);
  for (int i=0; i<L; ++i)
    tmp[i] = V*std::pow(i-(L-1.0)/2.0, 2);
  return tmp;
}
