#include "hip/hip_runtime.h"
#include <iostream>
#include "./include/Hamiltonian.hpp"
#include "./include/argparse.hpp"
#include "./include/magma_eigen_solver.cuh"

int main(int argc, char * argv[])
{
  std::vector<pair_t> options, defaults;
  // env; explanation of env
  options.push_back(pair_t("lx", "width of x-direction"));
  options.push_back(pair_t("ly", "width of y-direction"));
  options.push_back(pair_t("np", "# of particle numbers"));
  options.push_back(pair_t("t", "hopping element"));
  options.push_back(pair_t("U", "onsite interaction"));
  options.push_back(pair_t("mu1", "chemical potential (spin up)"));
  options.push_back(pair_t("mu2", "chemical potential (spin down)"));
  // env; default value
  defaults.push_back(pair_t("t", "1"));
  // parser for arg list
  argsparse parser(argc, argv, options, defaults);
  const unsigned int lx = parser.find<unsigned int>("lx"),
    ly = parser.find<unsigned int>("ly");
  const unsigned int nsites = lx*ly,
    nparticles = parser.find<unsigned int>("np");
  const unsigned int HilbertSize = ::combination(2*nsites, nparticles);
  const double t = parser.find<double>("t"),
    U = parser.find<double>("U"),
    mu1 = parser.find<double>("mu1"),
    mu2 = parser.find<double>("mu2");
  EigenSparseMatrix smatrix(HilbertSize);
  SquareLattice lattice(lx, ly);
  HubbardHamiltonian Hamiltonian(nsites, nparticles);
  std::cout << "# construct sparse matrix... " << std::flush;
  Hamiltonian.construct_matrix(smatrix, lattice, t, U, mu1, mu2);
  std::cout << "done." << std::endl << std::flush;

  auto & rawData = smatrix.get_data();
  rawData.prune(1e-15);
  rawData.makeCompressed();

  int * col = rawData.innerIndexPtr(),
    * row = rawData.outerIndexPtr();
  double * val = rawData.valuePtr();

  magma_init();
  magma_dopts opts;
  magma_queue_t queue;
  magma_queue_create(0, &queue);

  MAGMA_DLOBPCG_WRAPPER solver(HilbertSize);
  solver.construct_CSR_format(row, col, val, queue);

  // options for DLOBPCG solver
  opts.solver_par.solver = Magma_LOBPCG;
  opts.solver_par.num_eigenvalues = 1; // number of eigenvalues you want to compute
  opts.solver_par.ev_length = HilbertSize;
  opts.solver_par.maxiter = 1000; // max number of iterations
  opts.solver_par.verbose = true;
  opts.solver_par.rtol = 1e-7;
  opts.solver_par.restart = 8;
  // options for preconditioner
  opts.precond_par.solver = Magma_JACOBI;
  opts.precond_par.levels = 0; // ILU(0) - no fill-in
  opts.precond_par.trisolver = Magma_CUSOLVE; //exact triangular solves

  magma_deigensolverinfo_init(&opts.solver_par, queue);

  solver.run(opts, queue);

  std::cout << "--- lowest energy:" << std::endl;
  for (int i=0; i<opts.solver_par.num_eigenvalues; ++i)
    std::cout << opts.solver_par.eigenvalues[i] << " ";
  std::cout << std::endl;

  solver.free(queue);

  magma_queue_destroy(queue);
  magma_finalize();

  return 0;
}
